#include <stdint.h>
#include <map>
#include <chrono>
#include <cstdlib>

#include "CUDASieve/cudasieve.hpp"

using namespace std::chrono;

const uint64_t DEFAULT_SIEVE_SIZE = 1000000;
const int RESET_CYCLE_COUNT = 1000;

const std::map<uint64_t, const int> resultsDictionary =
{
    {          10UL, 4         },               // Historical data for validating our results - the number of primes
    {         100UL, 25        },               // to be found under some limit, such as 168 primes under 1000
    {        1000UL, 168       },
    {       10000UL, 1229      },
    {      100000UL, 9592      },
    {     1000000UL, 78498     },
    {    10000000UL, 664579    },
    {   100000000UL, 5761455   },
    {  1000000000UL, 50847534  },
    { 10000000000UL, 455052511 },
};

// Assumes any first argument is the desired sieve size. Defaults to DEFAULT_SIEVE_SIZE.
uint64_t determineSieveSize(int argc, char *argv[])
{
    if (argc < 2)
        return DEFAULT_SIEVE_SIZE;

    uint64_t sieveSize = strtoul(argv[1], nullptr, 0);

    if (sieveSize == 0) 
        return DEFAULT_SIEVE_SIZE;

    if (resultsDictionary.find(sieveSize) == resultsDictionary.end())
        fprintf(stderr, "WARNING: Results cannot be validated for selected sieve size of %zu!\n\n", sieveSize);
    
    return sieveSize;
}

void printResults(uint64_t sieveSize, size_t primeCount, double duration, uint64_t passes)
{
    auto expectedCount = resultsDictionary.find(sieveSize);
    auto countValidated = expectedCount != resultsDictionary.end() && expectedCount->second == primeCount;

    fprintf(stderr, "Passes: %zu, Time: %lf, Avg: %lf, Limit: %zu, Count: %zu, Validated: %d\n\n", 
            passes,
            duration,
            duration / passes,
            sieveSize,
            primeCount,
            countValidated);

    printf("rbergen_cuda;%zu;%f;1;algorithm=other,faithful=yes,bits=1\n", passes, duration);
}

int main(int argc, char *argv[])
{
    uint64_t sieveSize = determineSieveSize(argc, argv);
    uint64_t passes = 0;
    int cycleCount = 0;
    auto tStart = steady_clock::now();
    size_t primeCount;

    while (true)
    {
        // Implementation is faithful because CudaSieve::getDevicePrimes creates and destroys a sieve class instance
        uint64_t *primes = CudaSieve::getDevicePrimes(0, sieveSize, primeCount);
        passes++;
        hipFree(primes);
        if (duration_cast<seconds>(steady_clock::now() - tStart).count() >= 5)
        {
            printResults(sieveSize, primeCount, duration_cast<microseconds>(steady_clock::now() - tStart).count() / 1000000.0, passes);
            break;
        }
        cycleCount++;
        // Reset the device every RESET_CYCLE_COUNT passes, as recommended in the CUDASieve README
        if (cycleCount == RESET_CYCLE_COUNT)
        {
            hipDeviceReset();
            cycleCount = 0;
        }
    } 
}

