#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <cmath>
#include <algorithm>
#include <chrono>
#include <map>
#include <hip/hip_runtime.h>

#include "primes.h"

using namespace std::chrono;

__global__ void initialize_buffer(uint64_t blockSize, uint64_t wordCount, sieve_t *sieve)
{
    const uint64_t startIndex = uint64_t(blockIdx.x) * blockSize;
    // Don't initialize beyond the end of the buffer
    const uint64_t endIndex = ullmin(startIndex + blockSize, wordCount);

    // Set all block words to all 1s
    for (uint64_t index = startIndex; index < endIndex; index++)
        sieve[index] = MAX_WORD_VALUE;
}

__global__ void unmark_multiples_threads(uint32_t primeCount, uint32_t *primes, uint64_t halfSize, uint32_t sizeSqrt, sieve_t *sieve)
{
    // We unmark every "MAX_THREADS"th prime's multiples, starting with our thread index
    for (uint32_t primeIndex = threadIdx.x; primeIndex < primeCount; primeIndex += MAX_THREADS) 
    {
        const uint32_t prime = primes[primeIndex];
        const uint64_t primeSquared = uint64_t(prime) * prime;

        // Unmark multiples starting at just beyond the square root of the sieve size or the square of the prime, 
        //   whichever is larger.
        uint64_t firstUnmarked = primeSquared > sizeSqrt ? primeSquared : ((sizeSqrt / prime + 1) * prime);
        // We're marking off odd multiples only, so make sure we start with one of those!
        if (!(firstUnmarked & 1))
            firstUnmarked += prime;

        for (uint64_t index = firstUnmarked >> 1; index <= halfSize; index += prime) 
            // Clear the bit in the word that corresponds to the last part of the index 
            atomicAnd(&sieve[WORD_INDEX(index)], ~(sieve_t(1) << BIT_INDEX(index)));
    }
}

__global__ void unmark_multiples_blocks(uint32_t primeCount, uint32_t *primes, uint64_t halfSize, uint32_t sizeSqrt, uint32_t maxBlockIndex, uint64_t blockSize, sieve_t *sieve)
{
    // Calculate the start and end of the block we need to work on, at buffer word boundaries. 
    //   Note that the first variable is a number in sieve space...
    uint64_t blockStart = uint64_t(blockIdx.x) * blockSize + sizeSqrt;
    //   ...and the second is an index in the sieve buffer (representing odd numbers only)
    const uint64_t lastIndex = (blockIdx.x == maxBlockIndex) ? halfSize : (((blockStart + blockSize) & SIEVE_WORD_MASK) >> 1) - 1;

    // If this is not the first block, we actually start at the beginning of the first block word
    if (blockIdx.x != 0)
        blockStart &= SIEVE_WORD_MASK;

    #ifdef DEBUG
    printf("  - block %d: blockStart = %" PRIu64 " (index %" PRIu64 "), lastIndex = %" PRIu64 ".\n", blockIdx.x, blockStart, (blockStart >> 1), lastIndex);
    #endif

    for (uint32_t primeIndex = 0; primeIndex < primeCount; primeIndex++)
    {
        const uint32_t prime = primes[primeIndex];
        const uint64_t primeSquared = uint64_t(prime) * prime;

        // Unmark multiples starting at just beyond the start of our block or the square of the prime, 
        //   whichever is larger.
        uint64_t firstUnmarked = primeSquared >= blockStart ? primeSquared : ((blockStart / prime + 1) * prime);
        // We're marking off odd multiples only, so make sure we start with one of those!
        if (!(firstUnmarked & 1))
            firstUnmarked += prime;

        #if ROLLING_LIMIT > 0
        if (prime <= ROLLING_LIMIT)
        {
            uint64_t index = firstUnmarked >> 1;
            if (index > lastIndex)
                continue;

            uint64_t wordIndex = WORD_INDEX(index);
            uint32_t bitIndex = BIT_INDEX(index);                
            sieve_t bitMask = 0;

            do
            {
                // Check if our bit index has moved past the current word's bits. If so...
                if (bitIndex > MAX_BIT_INDEX) 
                {
                    // ...clear the current word's bits that are set in the mask, and move on the next word.
                    sieve[wordIndex++] &= ~bitMask;
                    // "Shift bitmask one word to the right" through calculation. It has to be done that way
                    //   in part because our word length may be the maximum the GPU supports (64 bits). 
                    bitIndex %= BITS_PER_WORD;
                    bitMask = sieve_t(1) << bitIndex;
                }
                else
                    // Just add the current bit index to the current word's mask
                    bitMask |= sieve_t(1) << bitIndex;

                // Add prime to overall sieve index and current word's bit index
                index += prime;
                bitIndex += prime;
            }
            while (index <= lastIndex);

            // Let's not forget to apply the last bitmask
            sieve[wordIndex] &= ~bitMask;
        }
        else
        {
        #endif // ROLLING_LIMIT > 0

            for (uint64_t index = firstUnmarked >> 1; index <= lastIndex; index += prime) 
                // Clear the bit in the word that corresponds to the last part of the index 
                sieve[WORD_INDEX(index)] &= ~(sieve_t(1) << BIT_INDEX(index));

        #if ROLLING_LIMIT > 0
        }
        #endif
    }
}

class Sieve 
{
    const uint64_t sieve_size;
    const uint64_t half_size;
    const uint32_t size_sqrt;
    const uint64_t buffer_word_size;
    const uint64_t buffer_byte_size;
    sieve_t *device_sieve_buffer;
    sieve_t *host_sieve_buffer;

    void unmark_multiples(Parallelization type, uint32_t primeCount, uint32_t *primeList) 
    {
        // Copy the first (square root of sieve size) buffer bytes to the device
        hipMemcpy(device_sieve_buffer, host_sieve_buffer, (size_sqrt >> 4) + 1, hipMemcpyHostToDevice);
        // Allocate device buffer for the list of primes and copy the prime list to it
        uint32_t *devicePrimeList;
        hipMalloc(&devicePrimeList, primeCount * sizeof(uint32_t));
        hipMemcpy(devicePrimeList, primeList, primeCount << 2, hipMemcpyHostToDevice);

        // Unmark multiples on the GPU using the selected method
        switch(type)
        {
            case Parallelization::threads:
            {
                // The number of threads we use is the maximum or the number of primes to process, whichever is lower
                const uint32_t threadCount = min(MAX_THREADS, primeCount);

                #ifdef DEBUG
                printf("- starting thread multiple unmarking with %u threads.\n", threadCount);
                #endif

                unmark_multiples_threads<<<1, threadCount>>>(primeCount, devicePrimeList, half_size, size_sqrt, device_sieve_buffer);
            }
            break;

            case Parallelization::blocks:
            {
                // Our workspace is the part of the sieve beyond the square root of its size...
                const uint64_t sieveSpace = sieve_size - size_sqrt;
                // ...which we halve and then divide by the word bit count to establish the number of words...
                uint64_t wordCount = sieveSpace >> (WORD_SHIFT + 1);
                // ...and increase that if the division left a remainder.
                if (sieveSpace & SIEVE_BITS_MASK)
                    wordCount++;
                
                // The number of blocks is the maximum thread count or the number of words, whichever is lower
                const uint32_t blockCount = (uint32_t)min(uint64_t(MAX_THREADS), wordCount);
                
                uint64_t blockSize = sieveSpace / blockCount;
                // Increase block size if the calculating division left a remainder
                if (sieveSpace % blockCount)
                    blockSize++;

                #ifdef DEBUG
                printf("- starting block multiple unmarking with blockCount %u and blockSize %zu.\n", blockCount, blockSize);
                #endif

                unmark_multiples_blocks<<<blockCount, 1>>>(primeCount, devicePrimeList, half_size, size_sqrt, blockCount - 1, blockSize, device_sieve_buffer);
            }
            break;

            default:
                // This is some method variation we don't know, so we warn and do nothing
                fprintf(stderr, "WARNING: Parallelization type %d unknown, multiple unmarking skipped!\n\n", to_underlying(type));
            break;
        }
        
        // Release the device prime list buffer
        hipFree(devicePrimeList);

        // Copy the sieve buffer from the device to the host. This function implies a wait for all GPU threads to finish.
        hipMemcpy(host_sieve_buffer, device_sieve_buffer, buffer_byte_size, hipMemcpyDeviceToHost);
        
        #ifdef DEBUG
        printf("- device to host copy of sieve buffer complete.\n");
        #endif
    }

    public:

    Sieve(unsigned long size) :
        sieve_size(size),
        half_size(size >> 1),
        size_sqrt((uint32_t)sqrt(size) + 1),
        buffer_word_size((half_size >> WORD_SHIFT) + 1),
        buffer_byte_size(buffer_word_size * BYTES_PER_WORD)
    {
        #ifdef DEBUG
        printf("- constructing sieve with buffer_word_size %zu and buffer_byte_size %zu.\n", buffer_word_size, buffer_byte_size);
        #endif

        // Allocate the device sieve buffer
        hipMalloc(&device_sieve_buffer, buffer_byte_size);

        // The number of blocks is the maximum number of threads or the number of words in the buffer, whichever is lower
        const uint32_t blockCount = (uint32_t)min(uint64_t(MAX_THREADS), buffer_word_size);
        
        uint64_t blockSize = buffer_word_size / blockCount;
        // Increase block size if the calculating division left a remainder
        if (buffer_word_size % blockCount)
            blockSize++;

        #ifdef DEBUG
        printf("- initializing device buffer with blockCount %u and blockSize %zu.\n", blockCount, blockSize);
        #endif

        initialize_buffer<<<blockCount, 1>>>(blockSize, buffer_word_size, device_sieve_buffer);

        // Allocate host sieve buffer (odd numbers only) and initialize the bytes up to the square root of the sieve 
        //   size to all 1s.
        host_sieve_buffer = (sieve_t *)malloc(buffer_byte_size);
        memset(host_sieve_buffer, 255, (size_sqrt >> 4) + 1);

        // Make sure the initialization of the device sieve buffer has completed
        hipDeviceSynchronize();

        #ifdef DEBUG
        printf("- post buffer initialization device sync complete.\n");
        #endif
    }

    ~Sieve() 
    {
        hipFree(device_sieve_buffer);
        free(host_sieve_buffer);
    }

    sieve_t *run(Parallelization type = Parallelization::threads)
    {
        // Calculate the size of the array we need to reserve for the primes we find up to and including the square root of
        //   the sieve size. x / (ln(x) - 1) is a good approximation, but often lower than the actual number, which would
        //   cause out-of-bound indexing. This is why we use x / (ln(x) - 1.2) to "responsibly over-allocate".
        const uint32_t primeListSize = uint32_t(double(size_sqrt) / (log(size_sqrt) - 1.2));

        uint32_t primeList[primeListSize];
        uint32_t primeCount = 0;

        // What follows is the basic Sieve of Eratosthenes algorithm, except we clear multiples up to and including the
        //   square root of the sieve size instead of to the sieve limit. We also keep track of the primes we find, so the
        //   GPU can unmark them later.
        const uint32_t lastMultipleIndex = size_sqrt >> 1;

        for (uint32_t factor = 3; factor <= size_sqrt; factor += 2)
        {
            uint64_t index = factor >> 1;

            if (host_sieve_buffer[WORD_INDEX(index)] & (sieve_t(1) << BIT_INDEX(index))) 
            {
                primeList[primeCount++] = factor;

                for (index = (factor * factor) >> 1; index <= lastMultipleIndex; index += factor)
                    host_sieve_buffer[WORD_INDEX(index)] &= ~(sieve_t(1) << BIT_INDEX(index));
            }
        }

        // Use the GPU to unmark the rest of the primes multiples
        unmark_multiples(type, primeCount, primeList);

        // Required to be truly compliant with Primes project rules
        return host_sieve_buffer;
    }

    uint64_t count_primes() 
    {
        uint64_t primeCount = 0;
        const uint64_t lastWord = WORD_INDEX(half_size);
        sieve_t word;

        // For all buffer words except the last one, just count the set bits in the word until there are none left.
        //   We only hold bits for odd numbers in the sieve buffer. However, due to a small "mathematical coincidence"
        //   bit 0 of word 0 effectively represents the only even prime 2. This means the "count set bits" approach 
        //   in itself yields the correct result.
        for (uint64_t index = 0; index < lastWord; index++)
        {
            word = host_sieve_buffer[index];
            while (word) 
            {
                if (word & 1)
                    primeCount++;

                word >>= 1;
            }
        }

        // For the last word, only count bits up to the (halved) sieve limit
        word = host_sieve_buffer[lastWord];
        const uint32_t lastBit = BIT_INDEX(half_size);
        for (uint32_t index = 0; word && index <= lastBit; index++) 
        {
            if (word & 1)
                primeCount++;
            
            word >>= 1;
        }

        return primeCount;
    }
};

const std::map<uint64_t, const int> resultsDictionary =
{
    {             10UL, 4         }, // Historical data for validating our results - the number of primes
    {            100UL, 25        }, //   to be found under some limit, such as 168 primes under 1000
    {          1'000UL, 168       },
    {         10'000UL, 1229      },
    {        100'000UL, 9592      },
    {      1'000'000UL, 78498     },
    {     10'000'000UL, 664579    },
    {    100'000'000UL, 5761455   },
    {  1'000'000'000UL, 50847534  },
    { 10'000'000'000UL, 455052511 },
};

const std::map<Parallelization, const char *> parallelizationDictionary = 
{
    { Parallelization::threads, "threads" },
    { Parallelization::blocks,  "blocks"  }
};

// Assumes any numerical first argument is the desired sieve size. Defaults to DEFAULT_SIEVE_SIZE.
uint64_t determineSieveSize(int argc, char *argv[])
{
    if (argc < 2)
        return DEFAULT_SIEVE_SIZE;

    const uint64_t sieveSize = strtoul(argv[1], nullptr, 0);

    if (sieveSize == 0) 
        return DEFAULT_SIEVE_SIZE;

    if (resultsDictionary.find(sieveSize) == resultsDictionary.end())
        fprintf(stderr, "WARNING: Results cannot be validated for selected sieve size of %zu!\n\n", sieveSize);
    
    return sieveSize;
}

void printResults(Parallelization type, uint64_t sieveSize, uint64_t primeCount, double duration, uint64_t passes)
{
    const auto expectedCount = resultsDictionary.find(sieveSize);
    const auto countValidated = expectedCount != resultsDictionary.end() && expectedCount->second == primeCount;
    const auto parallelizationEntry = parallelizationDictionary.find(type);
    const char *parallelizationLabel = parallelizationEntry != parallelizationDictionary.end() ? parallelizationEntry->second : "unknown";

    fprintf(stderr, "Passes: %zu, Time: %lf, Avg: %lf, Word size: %d, Max GPU threads: %d, Type: %s, Limit: %zu, Count: %zu, Validated: %d\n", 
            passes,
            duration,
            duration / passes,
            BITS_PER_WORD,
            MAX_THREADS,
            parallelizationLabel,
            sieveSize,
            primeCount,
            countValidated);

    printf("rbergen_faithful_cuda_%s;%zu;%f;1;algorithm=base,faithful=yes,bits=1\n\n", parallelizationLabel, passes, duration);
}

int main(int argc, char *argv[])
{
    const uint64_t sieveSize = determineSieveSize(argc, argv);

    Parallelization types[] = { Parallelization::blocks, Parallelization::threads };

    for (auto &type : types)
    {
        uint64_t passes = 0;

        Sieve *sieve = nullptr;

        const auto startTime = steady_clock::now();
        duration<double, std::micro> runTime;

        #ifndef DEBUG
        do
        {
        #endif

            delete sieve;

            sieve = new Sieve(sieveSize);
            sieve->run(type);

            passes++;

            runTime = steady_clock::now() - startTime;

        #ifndef DEBUG
        }
        while (duration_cast<seconds>(runTime).count() < 5);
        #endif
        
        #ifdef DEBUG
        printf("\n");
        #endif

        const size_t primeCount = sieve->count_primes();
        
        delete sieve;

        printResults(type, sieveSize, primeCount, duration_cast<microseconds>(runTime).count() / 1000000.0, passes); 
    }
}
